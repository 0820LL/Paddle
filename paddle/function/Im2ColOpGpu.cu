#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "Im2Col.h"

namespace paddle {

template<class T>
__global__
void im2colOCF(const T* imData, T* colData,
               int inputChannels,
               int inputHeight, int inputWidth,
               int filterHeight, int filterWidth,
               int strideHeight, int strideWidth,
               int paddingHeight, int paddingWidth,
               int outputHeight, int outputWidth) {
  int idx = threadIdx.x;
  int idy = threadIdx.y;
  int swId = blockIdx.x;
  int shId = blockIdx.y;

  for (int channelId = threadIdx.z;
       channelId < inputChannels;
       channelId += blockDim.z) {
    int widthOffset = idx + swId * strideWidth - paddingWidth;
    int heightOffset = idy + shId * strideHeight - paddingHeight;
    int imOffset = widthOffset + heightOffset * inputWidth
       + channelId * inputHeight * inputWidth;

    int colOffset = idx + idy * filterWidth
      + channelId * filterHeight * filterWidth
      + (shId * outputWidth + swId)
      * (inputChannels * filterHeight * filterWidth);

    if (idx < filterWidth && idy < filterHeight) {
      if (heightOffset >= inputHeight || heightOffset < 0 ||
          widthOffset >= inputWidth || widthOffset < 0) {
        colData[colOffset] = T(0);
      } else {
        colData[colOffset] = imData[imOffset];
      }
    }
  }
}

/*
 * imShape = [inputChannels, inputHeight, inputWidth]
 * colShape =
 *   [outputHeight, outputWidth, inputChannels, filterHeight, filterWidth]
 */
template <class T>
class Im2ColFunctor<kOCF, DEVICE_TYPE_GPU, T> {
public:
  void operator()(const T* imData,
                  const TensorShape& imShape,
                  T* colData,
                  const TensorShape& colShape,
                  int strideHeight,
                  int strideWidth,
                  int paddingHeight,
                  int paddingWidth) {
    int inputChannels = imShape[0];
    int inputHeight = imShape[1];
    int inputWidth = imShape[2];
    int filterHeight = colShape[3];
    int filterWidth = colShape[4];
    int outputHeight = colShape[0];
    int outputWidth = colShape[1];

    int blockDimX = 0;
    int blockDimY = 0;
    if (filterHeight <= 4 && filterWidth <= 4) {
      blockDimX = 4;
      blockDimY = 4;
    } else if (filterHeight <= 8 && filterWidth <= 8) {
      blockDimX = 8;
      blockDimY = 8;
    } else if (filterHeight <= 16 && filterWidth <= 16) {
      blockDimX = 16;
      blockDimY = 16;
    } else {
      blockDimX = 32;
      blockDimY = 32;
    }

    int blockDimZ = 1024 / blockDimX / blockDimY;
    dim3 threads(blockDimX, blockDimY, std::min(blockDimZ, inputChannels));
    dim3 grid(outputWidth, outputHeight);
    im2colOCF<T><<< grid, threads, 0, STREAM_DEFAULT >>>
        (imData, colData, inputChannels, inputHeight, inputWidth,
         filterHeight, filterWidth, strideHeight, strideWidth,
         paddingHeight, paddingWidth, outputHeight, outputWidth);
    CHECK_SYNC("Im2ColFunctor GPU failed");
  }
};

/*
 * imShape = [inputChannels, inputHeight, inputWidth]
 * colShape =
 *   [outputHeight, outputWidth, inputChannels, filterHeight, filterWidth]
 */
template <class T>
class Col2ImFunctor<kOCF, DEVICE_TYPE_GPU, T> {
public:
  void operator()(T* imData,
                  const TensorShape& imShape,
                  const T* colData,
                  const TensorShape& colShape,
                  int strideHeight,
                  int strideWidth,
                  int paddingHeight,
                  int paddingWidth) {
  }
};

template class Im2ColFunctor<kOCF, DEVICE_TYPE_GPU, float>;
template class Im2ColFunctor<kOCF, DEVICE_TYPE_GPU, double>;

}  // namespace paddle
