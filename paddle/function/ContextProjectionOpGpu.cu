#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "hl_base.h"
#include "ContextProjectionOp.h"

namespace paddle {

template <bool padding>
__global__ void KeContextProjectionForward(const real* input,
                                           const int* sequence,
                                           const real* weight,
                                           real* output,
                                           int input_dim,
                                           int context_length,
                                           int context_start,
                                           int begin_pad) {
  int idx = threadIdx.x;
  int block_size = blockDim.x;
  int sequenceId = blockIdx.x;
  int seq_start = sequence[sequenceId];
  int seq_end = sequence[sequenceId+1];
  real value = 0;

  int instances = seq_end - seq_start + context_length - 1;
  output += seq_start * input_dim * context_length;
  input += seq_start * input_dim;
  for (int k = 0; k <= input_dim / block_size; k++) {
    if (idx < input_dim) {
      for (int i = 0; i < instances; i++) {
        // i + context_start;
        if ((i + context_start) < 0) {
          if (padding) {
            value = weight[i * input_dim + idx];
          } else {
            continue;
          }
        } else if ((i + context_start) >= (seq_end - seq_start)) {
          if (padding) {
            value =
              weight[(begin_pad + i + context_start - (seq_end - seq_start)) *
                         input_dim + idx];
          } else {
            continue;
          }
        } else {
          value = input[(i + context_start) * input_dim + idx];
        }

        int outx = (i - context_length) < 0 ? i : (context_length - 1);
        int outy = (i - context_length) < 0 ? 0 : (i - (context_length - 1));
        real* output_r =
          output + outy * input_dim * context_length + outx * input_dim;
        for (int j = outy; j < seq_end - seq_start; j++) {
          output_r[idx] += value;
          if (j - outy == outx) break;
          output_r += (context_length - 1) * input_dim;
        }
      }
    }
    idx += block_size;
  }
}

void hl_context_projection_forward(const real* input,
                                   const int* sequence,
                                   real* weight,
                                   real* output,
                                   int num_sequences,
                                   int input_dim,
                                   int context_length,
                                   int context_start,
                                   int begin_pad,
                                   bool is_padding) {
  CHECK_NOTNULL(input);
  CHECK_NOTNULL(sequence);
  CHECK_NOTNULL(output);
  CHECK(!is_padding || weight);

  int block_size = 128;
  int blocks_x = num_sequences;
  int blocks_y = 1;
  dim3 threads(block_size, 1);
  dim3 grid(blocks_x, blocks_y);

  if (is_padding) {
    KeContextProjectionForward<true><<< grid, threads, 0, STREAM_DEFAULT >>>
      (input, sequence, weight, output, input_dim,
       context_length, context_start, begin_pad);
  } else  {
    KeContextProjectionForward<false><<< grid, threads, 0, STREAM_DEFAULT >>>
      (input, sequence, weight, output, input_dim,
       context_length, context_start, begin_pad);
  }
  CHECK_SYNC("hl_context_projection_forward failed");
}

template <>
void ContextProjectionForward<DEVICE_TYPE_GPU>(Tensor& output,
                                               const Tensor& input,
                                               const Tensor& weight,
                                               const Tensor& sequence,
                                               size_t context_length,
                                               int context_start,
                                               size_t begin_pad,
                                               bool is_padding) {
  CHECK(output.getData() && input.getData() && sequence.getData());
  CHECK_EQ(output.dims_.size(), 2);
  CHECK_EQ(input.dims_.size(), 2);
  CHECK_EQ(weight.dims_.size(), 2);
  CHECK_EQ(sequence.dims_.size(), 1);
  CHECK_EQ(output.dims_[1], input.dims_[1] * context_length);

  hl_context_projection_forward(input.getData(),
                                reinterpret_cast<int*>(sequence.getData()),
                                weight.getData(),
                                output.getData(),
                                sequence.dims_[0] - 1,
                                input.dims_[1],
                                context_length,
                                context_start,
                                begin_pad,
                                is_padding);
}

__global__ void KeContextProjectionBackwardData(real* out_grad,
                                                const int* sequence,
                                                real* in_grad,
                                                int input_dim,
                                                int context_length,
                                                int context_start) {
  int idx = threadIdx.x;
  int block_size = blockDim.x;
  int sequenceId = blockIdx.x;
  int seq_start = sequence[sequenceId];
  int seq_end = sequence[sequenceId+1];
  real value = 0;

  int instances = seq_end - seq_start + context_length - 1;
  out_grad += seq_start * input_dim * context_length;
  in_grad += seq_start * input_dim;
  for (int k = 0; k <= input_dim / block_size; k++) {
    if (idx < input_dim) {
      for (int i = 0; i < instances; i++) {
        if ((i + context_start) < 0) {
          continue;
        } else if ((i + context_start) >= (seq_end - seq_start)) {
          continue;
        } else {
          // value = 0;
          value = in_grad[(i + context_start) * input_dim + idx];
        }

        int outx = (i - context_length) < 0 ? i : (context_length - 1);
        int outy = (i - context_length) < 0 ? 0 : (i - (context_length - 1));
        real* output_r =
          out_grad + outy * input_dim * context_length + outx * input_dim;
        for (int j = outy; j < seq_end - seq_start; j++) {
          value += output_r[idx];
          if (j - outy == outx) break;
          output_r += (context_length - 1) * input_dim;
        }
        in_grad[(i + context_start) * input_dim + idx] = value;
      }
    }
    idx += block_size;
  }
}

void hl_context_projection_backward_data(real* out_grad,
                                         const int* sequence,
                                         real* input_grad,
                                         int num_sequences,
                                         int input_dim,
                                         int context_length,
                                         int context_start) {
  CHECK_NOTNULL(out_grad);
  CHECK_NOTNULL(sequence);
  CHECK_NOTNULL(input_grad);

  int block_size = 128;
  int blocks_x = num_sequences;
  int blocks_y = 1;
  dim3 threads(block_size, 1);
  dim3 grid(blocks_x, blocks_y);
  KeContextProjectionBackwardData<<< grid, threads, 0, STREAM_DEFAULT >>>
    (out_grad, sequence, input_grad, input_dim, context_length, context_start);
  CHECK_SYNC("hl_context_projection_backward_data failed");
}

template <>
void ContextProjectionBackwardData<DEVICE_TYPE_GPU>(Tensor& out_grad,
                                               Tensor& in_grad,
                                               const Tensor& sequence,
                                               size_t context_length,
                                               int context_start) {
  CHECK(in_grad.getData() && out_grad.getData() && sequence.getData());
  CHECK_EQ(out_grad.dims_.size(), 2);
  CHECK_EQ(in_grad.dims_.size(), 2);
  CHECK_EQ(sequence.dims_.size(), 1);
  CHECK_EQ(out_grad.dims_[1], in_grad.dims_[1] * context_length);

  hl_context_projection_backward_data(out_grad.getData(),
                reinterpret_cast<int*>(sequence.getData()),
                                      in_grad.getData(),
                                      sequence.dims_[0] - 1,
                                      in_grad.dims_[1],
                                      context_length,
                                      context_start);
}

template<int THREADS_X, int THREADS_Y>
__global__ void KeContextProjectionBackwardWeight(real* out_grad,
                                                  const int* sequence,
                                                  real* w_grad,
                                                  int num_sequences,
                                                  int w_dim,
                                                  int context_length,
                                                  int context_start,
                                                  int begin_pad) {
  __shared__ real sum_s[THREADS_Y][THREADS_X];
  int pad_of_block = (w_dim + THREADS_X - 1) / THREADS_X;
  const int idx = threadIdx.x;
  const int idy = threadIdx.y;
  int padId = blockIdx.x / pad_of_block;
  int weight_idx = idx + THREADS_X * (blockIdx.x % pad_of_block);
  int instanceId;
  real value = 0;
  real* output_r;

  sum_s[idy][idx] = 0.0f;
  if (weight_idx < w_dim) {
    for (int seqId = idy; seqId < num_sequences; seqId += THREADS_Y) {
      int seq_start = sequence[seqId];
      int seq_end = sequence[seqId+1];
      output_r = out_grad + seq_start * w_dim * context_length;

      if (context_start < 0) {
        if (padId + context_start < 0) {
          instanceId = padId;
        } else {
          // begin_pad > 0;
          instanceId = (padId - begin_pad) +
            (seq_end - seq_start) - context_start;
        }
      } else {
        if (padId + (seq_end - seq_start) < context_start) {
          continue;
        } else {
          // begin_pad == 0;
          instanceId = padId + (seq_end - seq_start) - context_start;
        }
      }

      int outx = (instanceId - context_length) < 0 ?
                 instanceId : (context_length - 1);
      int outy = (instanceId - context_length) < 0 ?
                 0 : (instanceId - (context_length - 1));
      output_r += outy * w_dim * context_length + outx * w_dim;
      for (int j = outy; j < seq_end - seq_start; j++) {
        value += output_r[weight_idx];
        if (j - outy == outx) break;
        output_r += (context_length - 1) * w_dim;
      }
    }
    sum_s[idy][idx] = value;
  }
  __syncthreads();

  for (int stride = THREADS_Y/2; stride > 0; stride = stride/2) {
    if (idy < stride) {
      sum_s[idy][idx] += sum_s[idy + stride][idx];
    }
    __syncthreads();
  }
  __syncthreads();

  if (weight_idx < w_dim) {
    if (idy == 0) {
      w_grad[padId * w_dim + weight_idx] += sum_s[0][idx];
    }
  }
}

void hl_context_projection_backward_weight(real* out_grad,
                                           const int* sequence,
                                           real* w_grad,
                                           int num_sequences,
                                           int w_dim,
                                           size_t total_pad,
                                           int context_length,
                                           int context_start,
                                           int begin_pad) {
  CHECK_NOTNULL(out_grad);
  CHECK_NOTNULL(sequence);
  CHECK_NOTNULL(w_grad);

  int threads_x = 32;
  int threads_y = 32;
  int blocks_x = total_pad * ((w_dim + threads_x - 1) / threads_x);
  dim3 threads(threads_x, threads_y);
  dim3 grid(blocks_x, 1);

  KeContextProjectionBackwardWeight<32, 32>
    <<< grid, threads, 0, STREAM_DEFAULT >>>
    (out_grad, sequence, w_grad, num_sequences, w_dim,
     context_length, context_start, begin_pad);
  CHECK_SYNC("hl_context_projection_backward_weight failed");
}

template <>
void ContextProjectionBackwardWeight<DEVICE_TYPE_GPU>(Tensor& out_grad,
                                                      Tensor& w_grad,
                                                      const Tensor& sequence,
                                                      size_t context_length,
                                                      int context_start,
                                                      size_t total_pad,
                                                      size_t begin_pad) {
  CHECK(w_grad.getData() && out_grad.getData() && sequence.getData());
  CHECK_EQ(out_grad.dims_.size(), 2);
  CHECK_EQ(w_grad.dims_.size(), 2);
  CHECK_EQ(sequence.dims_.size(), 1);
  CHECK_EQ(out_grad.dims_[1], w_grad.dims_[1] * context_length);

  hl_context_projection_backward_weight(out_grad.getData(),
                    reinterpret_cast<int*>(sequence.getData()),
                                        w_grad.getData(),
                                        sequence.dims_[0] - 1,
                                        w_grad.dims_[1],
                                        total_pad,
                                        context_length,
                                        context_start,
                                        begin_pad);
}

template <>
void ContextProjectionBackward<DEVICE_TYPE_GPU>(Tensor& out_grad,
                                               Tensor& in_grad,
                                               Tensor& w_grad,
                                               const Tensor& sequence,
                                               size_t context_length,
                                               int context_start,
                                               size_t begin_pad,
                                               bool is_padding,
                                               size_t total_pad) {
    if (in_grad.getData()) {
        ContextProjectionBackwardData<DEVICE_TYPE_GPU>(out_grad,
                in_grad,
                sequence,
                context_length,
                context_start);
    }
    if (is_padding && w_grad.getData()) {
        ContextProjectionBackwardWeight<DEVICE_TYPE_GPU>(out_grad,
                w_grad,
                sequence,
                context_length,
                context_start,
                total_pad,
                begin_pad);
  }
}

}  // namespace paddle
