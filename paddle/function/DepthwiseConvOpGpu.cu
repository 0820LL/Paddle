#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "ConvOp.h"
#include "DepthwiseConvOp.h"

namespace paddle {
template <class T>
__global__ void ConvolutionDepthwiseWeightForward(const int nthreads,
    const T* const bottom_data, const T* const weight_data,
    const int num, const int channels, const int top_height,
    const int top_width, const int bottom_height, const int bottom_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w,
    const int dilation_h, const int dilation_w, T* const top_data) {

  int index =
    (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  
  if(index < nthreads) {
    const int n = index / channels / top_height / top_width;
    const int c = (index / top_height / top_width) % channels;
    const int h = (index / top_width) % top_height;
    const int w = index % top_width;
    const T* weight = weight_data + c * kernel_h * kernel_w;
    T value = 0;
    for (int kh = 0; kh < kernel_h; ++kh) {
      for (int kw = 0; kw < kernel_w; ++kw) {
        const int h_in = -pad_h + h * stride_h + kh * dilation_h;
        const int w_in = -pad_w + w * stride_w + kw * dilation_w;
        if ((h_in >= 0) && (h_in < bottom_height)
              && (w_in >= 0) && (w_in < bottom_width)) {
          const int offset = ((n * channels + c) * bottom_height + h_in)
                * bottom_width + w_in;
          value += (*weight) * bottom_data[offset];
        }
        ++weight;
      }
    }
    top_data[index] = value;
  }
}

template <class T>
__global__ void ConvolutionDepthwiseBottomBackward(const int nthreads,
    const T* const top_diff, const T* const weight_data,
    const int num, const int channels, const int top_height,
    const int top_width, const int bottom_height, const int bottom_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w,
    const int dilation_h, const int dilation_w, T* const bottom_diff) {
  int index =
    (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  if(index < nthreads) {
    const int n = index / channels / bottom_height / bottom_width;
    const int c = (index / bottom_height / bottom_width) % channels;
    const int h = (index / bottom_width) % bottom_height;
    const int w = index % bottom_width;
    const T* weight = weight_data + c * kernel_h * kernel_w;
    T value = 0;
    for (int kh = 0; kh < kernel_h; ++kh) {
      for (int kw = 0; kw < kernel_w; ++kw) {
        const int h_out_s = h + pad_h - kh * dilation_h;
        const int w_out_s = w + pad_w - kw * dilation_w;
        if (((h_out_s % stride_h) == 0) && ((w_out_s % stride_w) == 0)) {
          const int h_out = h_out_s / stride_h;
          const int w_out = w_out_s / stride_w;
	  //it affect the effectives
          if ((h_out >= 0) && (h_out < top_height)
                && (w_out >= 0) && (w_out < top_width)) {
            const int offset = ((n * channels + c) * top_height + h_out)
                  * top_width + w_out;
            value += (*weight) * top_diff[offset];
          }
        }
        ++weight;
      }
    }
    bottom_diff[index] += value;
  }
}

template <class T>
__global__ void ConvolutionDepthwiseWeightBackward(const int num_i, const int nthreads,
    const T* const top_diff, const T* const bottom_data,
    const int num, const int channels, const int top_height,
    const int top_width, const int bottom_height, const int bottom_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w,
    const int dilation_h, const int dilation_w, T* const buffer_data) {
  int index =
    (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  if (index < nthreads) {
    const int h = (index / top_width) % top_height;
    const int w = index % top_width;
    const int kh = (index / kernel_w / top_height / top_width)
          % kernel_h;
    const int kw = (index / top_height / top_width) % kernel_w;
    const int h_in = -pad_h + h * stride_h + kh * dilation_h;
    const int w_in = -pad_w + w * stride_w + kw * dilation_w;
    if ((h_in >= 0) && (h_in < bottom_height)
          && (w_in >= 0) && (w_in < bottom_width)) {
      const int c = index / kernel_h / kernel_w / top_height / top_width;
      const int n = num_i;
      const int top_offset = ((n * channels + c) * top_height + h)
            * top_width + w;
      const int bottom_offset = ((n * channels + c) * bottom_height + h_in)
            * bottom_width + w_in;
      buffer_data[index] = top_diff[top_offset] * bottom_data[bottom_offset];
    } else {
      buffer_data[index] = 0;
    }
  }
}

template <class T>
class DepthwiseConvFunctor<DEVICE_TYPE_GPU, T>{
public:
  void operator()(int outputSize, 
            const T* inputData, 
            const T* filterData,
            int batchSize,
            int outputChannels,
            int outputHeight,
            int outputWidth,
            int filterHeight,
            int filterWidth,
            int strideH,
            int strideW,
            int paddingH,
            int paddingW,
            T* outputData){

    size_t blocks = (outputSize + 1024 -1) / 1024;
    size_t blockX = 512;
    size_t blockY = (blocks+512-1)/512;
    dim3 threads(1024, 1);
    dim3 grid(blockX, blockY);
    
    ConvolutionDepthwiseWeightForward<T>
        <<< grid, threads, 0, STREAM_DEFAULT >>>(
            outputSize, 
            inputData, 
            filterData,
            batchSize,
            outputChannels,
            outputHeight,
            outputWidth,
            filterHeight,
            filterWidth,
            strideH,
            strideW,
            paddingH,
            paddingW,
            outputData);
    }
};

template <class T>
class DepthwiseConvGradInputFunctor<DEVICE_TYPE_GPU, T>{
public:
  void operator()(int inputSize,
            const T* outputGrad,
            const T* filterData,
            int batchSize,
            int outputChannels,
            int outputHeight,
            int outputWidth,
            int inputHeight,
            int inputWidth,
            int filterHeight,
            int filterWidth,
            int strideH,
            int strideW,
            int paddingH,
            int paddingW,
                T* inputGrad){

    size_t blocks = (inputSize + 1024 -1) / 1024;
    size_t blockX = 512;
    size_t blockY = (blocks+512-1)/512;
    dim3 threads(1024, 1);
    dim3 grid(blockX, blockY);

    ConvolutionDepthwiseBottomBackward<T>
          // NOLINT_NEXT_LINE(whitespace/operators)
        <<< grid, threads, 0, STREAM_DEFAULT >>>(
            inputSize,
            outputGrad,
            filterData,
            batchSize,
            outputChannels,
            outputHeight,
            outputWidth,
            inputHeight,
            inputWidth,
            filterHeight,
            filterWidth,
            strideH,
            strideW,
            paddingH,
            paddingW,
            inputGrad);
    }
};

template <class T>
class DepthwiseConvGradFilterFunctor<DEVICE_TYPE_GPU, T> {
public:
  void operator()(int num_i,
                int colDataSize,
                const T* outputGrad,
                const T* inputData,
                int batchSize,
                int outputChannels,
                int outputHeight,
                int outputWidth,
                int inputHeight,
                int inputWidth,
                int filterHeight,
                int filterWidth,
                int strideH,
                int strideW,
                int paddingH,
                int paddingW,
                T* colData,
                T* multiplierData,
                T* filterGrad){

        size_t blocks = (colDataSize + 1024 -1) / 1024;
        size_t blockX = 512;
        size_t blockY = (blocks+512-1)/512;
        dim3 threads(1024, 1);
        dim3 grid(blockX, blockY);

	    ConvolutionDepthwiseWeightBackward<T>
            <<< grid, threads, 0, STREAM_DEFAULT >>>(
                i,
                size,
                outputGrad,
                inputData,
                batchSize,
                outputChannels,
                outputHeight,
                outputWidth,
                inputHeight,
                inputWidth,
                filterHeight,
                filterWidth,
                strideH,
                strideW,
                paddingH,
                paddingW,
                colData
            );
        GemmFunctor<Device, real> gemm;
        int M = size / outputHeight / outputWidth;
        int N = 1;
        int K = outputHeight * outputWidth;
        gemm(CblasNoTrans,
            CblasNoTrans,
            M,
            N,
            K,
            1.0f,
            colData,
            K,
            multiplierData,
            N,
            1.0f,
            filterGrad,
            N);
        //gemv
    }
};

template class DepthwiseConvGradInputFunctor<DEVICE_TYPE_GPU, float>;
template class DepthwiseConvGradInputFunctor<DEVICE_TYPE_GPU, double>;
template class DepthwiseConvFunctor<DEVICE_TYPE_GPU, float>;
template class DepthwiseConvFunctor<DEVICE_TYPE_GPU, double>;
template class DepthwiseConvGradFilterFunctor<DEVICE_TYPE_GPU, float>;
template class DepthwiseConvGradFilterFunctor<DEVICE_TYPE_GPU, double>;

}  // namespace paddle
