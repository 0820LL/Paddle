#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "DepthwiseConvOp.h"
#include "GemmFunctor.h"
#include "paddle/math/BaseMatrix.h"

namespace paddle {
// CUDA kernel to compute the depthwise convolution forward pass
template <class T>
__global__ 
void ConvolutionDepthwiseForward(const int nthreads,
    const T* const inputData, const T* const filterData,
    const int batchSize, const int outputChannels, const int outputHeight,
    const int outputWidth, const int inputHeight, const int inputWidth,
    const int filterHeight, const int filterWidth, const int strideH,
    const int strideW, const int paddingH, const int paddingW,
    T* const outputData) {

  int index =
    (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  
  if(index < nthreads) {
    const int n = index / outputChannels / outputHeight / outputWidth;
    const int c = (index / outputHeight / outputWidth) % outputChannels;
    const int h = (index / outputWidth) % outputHeight;
    const int w = index % outputWidth;
    const T* weight = filterData + c * filterHeight * filterWidth;
    T value = 0;
	const int h_in_start = -paddingH + h * strideH;
	const int w_in_start = -paddingW + w * strideW;
	const int h_in_end = -paddingH + h * strideH + filterHeight - 1;
	const int w_in_end = -paddingW + w * strideW + filterWidth - 1;
    if ((h_in_start >= 0) && (h_in_end < inputHeight) 
		 &&(w_in_start >= 0) && (w_in_end < inputWidth)) {
		for (int kh = 0; kh < filterHeight; ++kh) {
		  for (int kw = 0; kw < filterWidth; ++kw) {
			const int h_in = -paddingH + h * strideH + kh;
			const int w_in = -paddingW + w * strideW + kw;
			const int offset = ((n * outputChannels + c) * inputHeight + h_in)
					* inputWidth + w_in;
			  value += (*weight) * inputData[offset];
			++weight;
		  }
		}
	}else{
		for (int kh = 0; kh < filterHeight; ++kh) {
		  for (int kw = 0; kw < filterWidth; ++kw) {
			const int h_in = -paddingH + h * strideH + kh;
			const int w_in = -paddingW + w * strideW + kw;
			if ((h_in >= 0) && (h_in < inputHeight)
				  && (w_in >= 0) && (w_in < inputWidth)) {
			  const int offset = ((n * outputChannels + c) * inputHeight + h_in)
					* inputWidth + w_in;
			  value += (*weight) * inputData[offset];
			}
			++weight;
		  }
		}
	}
    outputData[index] = value;
  }
}

// CUDA kernel to compute the depthwise convolution backprop w.r.t input.
template <class T>
__global__
void ConvolutionDepthwiseInputBackward(const int nthreads,
    const T* const top_diff, const T* const weight_data,
    const int num, const int outputChannels, const int outputHeight,
    const int outputWidth, const int inputHeight, const int inputWidth,
    const int filterHeight, const int filterWidth, const int strideH,
    const int strideW, const int paddingH, const int paddingW,
     T* const bottom_diff) {
  int index =
    (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  if(index < nthreads) {
    const int n = index / outputChannels / inputHeight / inputWidth;
    const int c = (index / inputHeight / inputWidth) % outputChannels;
    const int h = (index / inputWidth) % inputHeight;
    const int w = index % inputWidth;
    const T* weight = weight_data + c * filterHeight * filterWidth;
    T value = 0;
    for (int kh = 0; kh < filterHeight; ++kh) {
      for (int kw = 0; kw < filterWidth; ++kw) {
        const int h_out_s = h + paddingH - kh;
        const int w_out_s = w + paddingW - kw;
        if (((h_out_s % strideH) == 0) && ((w_out_s % strideW) == 0)) {
          const int h_out = h_out_s / strideH;
          const int w_out = w_out_s / strideW;
	     // TODO(zhaolong) : the 'if' affect the effectiveness, it needs to optimize
          if ((h_out >= 0) && (h_out < outputHeight)
                && (w_out >= 0) && (w_out < outputWidth)) {
            const int offset = ((n * outputChannels + c) * outputHeight + h_out)
                  * outputWidth + w_out;
            value += (*weight) * top_diff[offset];
          }
        }
        ++weight;
      }
    }
    bottom_diff[index] += value;
  }
}

// CUDA kernel to compute the depthwise convolution backprop w.r.t filter.
template <class T>
__global__
void ConvolutionDepthwiseFilterBackward(const int num_i, const int nthreads,
    const T* const top_diff, const T* const inputData,
    const int num, const int outputChannels, const int outputHeight,
    const int outputWidth, const int inputHeight, const int inputWidth,
    const int filterHeight, const int filterWidth, const int strideH,
    const int strideW, const int paddingH, const int paddingW,
    T* const buffer_data) {
  int index =
    (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  if (index < nthreads) {
    const int h = (index / outputWidth) % outputHeight;
    const int w = index % outputWidth;
    const int kh = (index / filterWidth / outputHeight / outputWidth)
          % filterHeight;
    const int kw = (index / outputHeight / outputWidth) % filterWidth;
    const int h_in = -paddingH + h * strideH + kh;
    const int w_in = -paddingW + w * strideW + kw;
    if ((h_in >= 0) && (h_in < inputHeight)
          && (w_in >= 0) && (w_in < inputWidth)) {
      const int c = index / filterHeight / filterWidth / outputHeight / outputWidth;
      const int n = num_i;
      const int top_offset = ((n * outputChannels + c) * outputHeight + h)
            * outputWidth + w;
      const int bottom_offset = ((n * outputChannels + c) * inputHeight + h_in)
            * inputWidth + w_in;
      buffer_data[index] = top_diff[top_offset] * inputData[bottom_offset];
    } else {
      buffer_data[index] = 0;
    }
  }
}

template <class T>
class DepthwiseConvFunctor<DEVICE_TYPE_GPU, T>{
public:
  void operator()(const T* inputData, 
            const T* filterData,
            int batchSize,
            int outputChannels,
            int outputHeight,
            int outputWidth,
            int inputHeight,
            int inputWidth,
            int filterHeight,
            int filterWidth,
            int strideH,
            int strideW,
            int paddingH,
            int paddingW,
            T* outputData){

    int outputSize = batchSize * outputChannels * outputHeight * outputWidth;

    size_t blocks = (outputSize + 1024 -1) / 1024;
    size_t blockX = 512;
    size_t blockY = (blocks+512-1)/512;
    dim3 threads(1024, 1);
    dim3 grid(blockX, blockY);

    ConvolutionDepthwiseForward<T>
        <<< grid, threads, 0, STREAM_DEFAULT >>>(
            outputSize, 
            inputData, 
            filterData,
            batchSize,
            outputChannels,
            outputHeight,
            outputWidth,
            inputHeight,
            inputWidth,
            filterHeight,
            filterWidth,
            strideH,
            strideW,
            paddingH,
            paddingW,
            outputData);
    }
};

template <class T>
class DepthwiseConvGradInputFunctor<DEVICE_TYPE_GPU, T>{
public:
  void operator()(const T* outputGrad,
            const T* filterData,
            int batchSize,
            int outputChannels,
            int outputHeight,
            int outputWidth,
            int inputChannels,
            int inputHeight,
            int inputWidth,
            int filterHeight,
            int filterWidth,
            int strideH,
            int strideW,
            int paddingH,
            int paddingW,
            T* inputGrad){

	int inputSize = batchSize * inputChannels * inputHeight * inputWidth;

    size_t blocks = (inputSize + 1024 -1) / 1024;
    size_t blockX = 512;
    size_t blockY = (blocks+512-1)/512;
    dim3 threads(1024, 1);
    dim3 grid(blockX, blockY);


    ConvolutionDepthwiseInputBackward<T>
          // NOLINT_NEXT_LINE(whitespace/operators)
        <<< grid, threads, 0, STREAM_DEFAULT >>>(
            inputSize,
            outputGrad,
            filterData,
            batchSize,
            outputChannels,
            outputHeight,
            outputWidth,
            inputHeight,
            inputWidth,
            filterHeight,
            filterWidth,
            strideH,
            strideW,
            paddingH,
            paddingW,
            inputGrad);
    }
};

template <class T>
class DepthwiseConvGradFilterFunctor<DEVICE_TYPE_GPU, T> {
public:
  void operator()(const T* outputGrad,
                const T* inputData,
                int batchSize,
                int outputChannels,
                int outputHeight,
                int outputWidth,
                int inputChannels,
                int inputHeight,
                int inputWidth,
                int filterHeight,
                int filterWidth,
                int strideH,
                int strideW,
                int paddingH,
                int paddingW,
                T* colData,
                T* filterGrad){

        int colDataSize = inputChannels * filterHeight * filterWidth * outputHeight * outputWidth;

        size_t blocks = (colDataSize + 1024 -1) / 1024;
        size_t blockX = 512;
        size_t blockY = (blocks+512-1)/512;
        dim3 threads(1024, 1);
        dim3 grid(blockX, blockY);
		BaseMatrix filterGradMatrix(inputChannels * filterHeight * filterWidth, 1, filterGrad, false, true);

        for(int i = 0; i < batchSize; i++) {
			ConvolutionDepthwiseFilterBackward<T>
				<<< grid, threads, 0, STREAM_DEFAULT >>>(
                    i,
                    colDataSize,
                    outputGrad,
                    inputData,
                    batchSize,
                    outputChannels,
                    outputHeight,
                    outputWidth,
                    inputHeight,
                    inputWidth,
                    filterHeight,
                    filterWidth,
                    strideH,
                    strideW,
                    paddingH,
                    paddingW,
                    colData
				);
			int M = colDataSize / outputHeight / outputWidth;
			int K = outputHeight * outputWidth;

            BaseMatrix colMatrix(M, K, colData, false, true);
		    filterGradMatrix.sumRows(colMatrix, (T)1.0, (T)1.0);	
		}
    }
};

#ifdef PADDLE_TYPE_DOUBLE
template class DepthwiseConvGradInputFunctor<DEVICE_TYPE_GPU, double>;
template class DepthwiseConvFunctor<DEVICE_TYPE_GPU, double>;
template class DepthwiseConvGradFilterFunctor<DEVICE_TYPE_GPU, double>;
#else 
template class DepthwiseConvGradInputFunctor<DEVICE_TYPE_GPU, float>;
template class DepthwiseConvFunctor<DEVICE_TYPE_GPU, float>;
template class DepthwiseConvGradFilterFunctor<DEVICE_TYPE_GPU, float>;
#endif

}  // namespace paddle
