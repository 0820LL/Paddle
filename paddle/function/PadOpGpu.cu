#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "hl_base.h"
#include "PadOp.h"

namespace paddle {

__global__ void KePad(real* outputs, const real* inputs,
                      int inC, int inH, int inW,
                      int padc, int padh, int padw,
                      int outC, int outH, int outW, int nthreads) {
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < nthreads) {
    const int w = idx % inW;
    const int h = (idx / inW) % inH;
    const int c = (idx / inW / inH) % inC;
    const int n = idx / inW / inH / inC;

    const int off = ((n * outC + c + padc) * outH + h + padh) * outW + padw + w;
    outputs[off] = inputs[idx];
  }
}

template <>
void Pad<DEVICE_TYPE_GPU>(real* outputs,
                          const real* inputs,
                          const int num,
                          const int inC,
                          const int inH,
                          const int inW,
                          const int padc0,
                          const int padc1,
                          const int padh0,
                          const int padh1,
                          const int padw0,
                          const int padw1) {
  size_t nth = num * inC * inH * inW;
  int blockSize = 1024;
  int gridSize = (nth + 1024 - 1) / 1024;
  int outC = inC + padc0 + padc1;
  int outH = inH + padh0 + padh1;
  int outW = inW + padw0 + padw1;
  KePad<<<gridSize, blockSize, 0, STREAM_DEFAULT>>>
    (outputs, inputs, inC, inH, inW, padc0, padh0, padw0,
     outC, outH, outW, nth);
  CHECK_SYNC("Pad");
}

__global__ void KePadDiff(real* inGrad, const real* outGrad,
                          int inC, int inH, int inW,
                          int padc, int padh, int padw,
                          int outC, int outH, int outW, int nthreads) {
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < nthreads) {
    const int w = idx % inW;
    const int h = (idx / inW) % inH;
    const int c = (idx / inW / inH) % inC;
    const int n = idx / inW / inH / inC;

    const int off = ((n * outC + c + padc) * outH + h + padh) * outW + padw + w;
    inGrad[idx] += outGrad[off];
  }
}

template <>
void PadGrad<DEVICE_TYPE_GPU>(real* inGrad,
                              const real* outGrad,
                              const int num,
                              const int inC,
                              const int inH,
                              const int inW,
                              const int padc0,
                              const int padc1,
                              const int padh0,
                              const int padh1,
                              const int padw0,
                              const int padw1) {
  int nth = num * inC * inH * inW;
  int blockSize = 1024;
  int gridSize = (nth + 1024 - 1) / 1024;
  int outC = inC + padc0 + padc1;
  int outH = inH + padh0 + padh1;
  int outW = inW + padw0 + padw1;
  KePadDiff <<<gridSize, blockSize, 0, STREAM_DEFAULT>>>
    (inGrad, outGrad, inC, inH, inW, padc0, padh0, padw0,
     outC, outH, outW, nth);
  CHECK_SYNC("PadGrad");
}

}  // namespace paddle
