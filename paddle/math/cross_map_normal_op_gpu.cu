#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "hl_base.h"
#include "cross_map_normal_op.h"

namespace paddle {

__global__ void KeCMRNormFillScale(size_t imageSize, const real* in,
                                   real* scale, size_t channels,
                                   size_t height, size_t width, size_t size,
                                   real alpha) {
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < imageSize) {
    const int w = idx % width;
    const int h = (idx / width) % height;
    const int n = idx / width / height;
    const int offset = (n * channels * height + h) * width + w;

    in += offset;
    scale += offset;
    const int step = height * width;
    const int pre_pad = (size - 1) / 2;
    const int post_pad = size - pre_pad - 1;

    real accum = 0;
    int index = 0;
    while (index < channels + post_pad) {
      if (index < channels) {
        accum += in[index * step] * in[index * step];
      }
      if (index >= size) {
        accum -= in[(index - size) * step] * in[(index - size) * step];
      }
      if (index >= post_pad) {
        scale[(index - post_pad) * step] = 1. + accum * alpha;
      }
      ++index;
    }
  }
}

__global__ void KeCMRNormOutput(size_t inputSize, const real* in,
                                const real* scale, real negative_beta,
                                real* out) {
  const int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < inputSize) {
    out[index] = in[index] * pow(scale[index], negative_beta);
  }
}

template <>
void CrossMapNormal<DEVICE_TYPE_GPU>::operator()(GpuMatrix& outputs,
                                                 GpuMatrix& denoms,
                                                 GpuMatrix& inputs,
                                                 size_t channels,
                                                 size_t imgSizeH,
                                                 size_t imgSizeW,
                                                 size_t sizeX,
                                                 real scale,
                                                 real pow) {
  CHECK(outputs.isContiguous());
  CHECK(inputs.isContiguous());
  CHECK(denoms.isContiguous());
  CHECK_EQ(outputs.getHeight(), inputs.getHeight());
  CHECK_EQ(outputs.getWidth(), inputs.getWidth());
  CHECK_EQ(outputs.getHeight(), denoms.getHeight());
  CHECK_EQ(outputs.getWidth(), denoms.getWidth());

  size_t numSample = inputs.getHeight();
  size_t numCols = inputs.getWidth();
  CHECK(imgSizeH * imgSizeW * channels == numCols);

  real* inputsData = inputs.getData();
  real* denomsData = denoms.getData();
  real* outputsData = outputs.getData();

  size_t imageSize = numSample * imgSizeH * imgSizeW;
  int blockSize = 1024;
  int gridSize = (imageSize + 1024 - 1) / 1024;
  KeCMRNormFillScale<<<gridSize, blockSize, 0, STREAM_DEFAULT>>>
    (imageSize, inputsData, denomsData,
    channels, imgSizeH, imgSizeW, sizeX, scale);

  size_t inputSize = numSample * imgSizeH * imgSizeW *channels;
  blockSize = 1024;
  gridSize = (inputSize + 1024 - 1) / 1024;
  KeCMRNormOutput<<<gridSize, blockSize, 0, STREAM_DEFAULT>>>
    (inputSize, inputsData, denomsData, -pow, outputsData);

  CHECK_SYNC("CrossMapNormalFwd");
}

__global__ void KeCMRNormDiff(size_t imageSize, const real* bottom_data,
                              const real* top_data, const real* scale,
                              const real* top_diff, size_t channels,
                              size_t height, size_t width, size_t size,
                              real negative_beta, real cache_ratio,
                              real* bottom_diff ) {
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < imageSize) {
    const int w = idx % width;
    const int h = (idx / width) % height;
    const int n = idx / width / height;
    const int offset = (n * channels * height + h) * width + w;
    bottom_data += offset;
    top_data += offset;
    scale += offset;
    top_diff += offset;
    bottom_diff += offset;

    const int step = height * width;
    const int pre_pad = size - (size + 1) / 2;
    const int post_pad = size - pre_pad - 1;

    int index = 0;
    real accum = 0;
    while (index < channels + post_pad) {
      if (index < channels) {
        accum += top_diff[index * step] * top_data[index * step] /
          scale[index * step];
      }
      if (index >= size) {
        accum -= top_diff[(index - size) * step] *
          top_data[(index - size) * step] / scale[(index - size) * step];
      }
      if (index >= post_pad) {
        bottom_diff[(index - post_pad) * step] +=
          top_diff[(index - post_pad) * step] *
          pow(scale[(index - post_pad) * step], negative_beta) - cache_ratio *
          bottom_data[(index - post_pad) * step] * accum;
      }
      ++index;
    }
  }
}

template <>
void CrossMapNormalGrad<DEVICE_TYPE_GPU>::operator()(GpuMatrix& inputsGrad,
                                                     GpuMatrix& inputsValue,
                                                     GpuMatrix& outputsGrad,
                                                     GpuMatrix& outputsValue,
                                                     GpuMatrix& denoms,
                                                     size_t channels,
                                                     size_t imgSizeH,
                                                     size_t imgSizeW,
                                                     size_t sizeX,
                                                     real scale,
                                                     real pow) {
  CHECK(inputsGrad.isContiguous());
  CHECK(outputsGrad.isContiguous());
  CHECK(denoms.isContiguous());
  CHECK(inputsValue.isContiguous());
  CHECK(outputsValue.isContiguous());
  CHECK_EQ(inputsGrad.getHeight(), outputsGrad.getHeight());
  CHECK_EQ(inputsGrad.getWidth(), outputsGrad.getWidth());
  CHECK_EQ(inputsGrad.getHeight(), denoms.getHeight());
  CHECK_EQ(inputsGrad.getWidth(), denoms.getWidth());
  CHECK_EQ(inputsGrad.getHeight(), inputsValue.getHeight());
  CHECK_EQ(inputsGrad.getWidth(), inputsValue.getWidth());
  CHECK_EQ(inputsGrad.getHeight(), outputsValue.getHeight());
  CHECK_EQ(inputsGrad.getWidth(), outputsValue.getWidth());

  size_t numSample = inputsGrad.getHeight();
  size_t numCols = inputsGrad.getWidth();
  CHECK(imgSizeH * imgSizeW * channels == numCols);

  size_t imageSize = numSample * imgSizeH * imgSizeW;
  real* inputsGradData = inputsGrad.getData();
  real* inputsData = inputsValue.getData();
  real* denomsData = denoms.getData();
  real* outputsGradData = outputsGrad.getData();
  real* outputsData = outputsValue.getData();

  int blockSize = 1024;
  int gridSize = (imageSize + 1024 - 1) / 1024;
  KeCMRNormDiff <<<gridSize, blockSize, 0, STREAM_DEFAULT>>>
    (imageSize, inputsData, outputsData, denomsData, outputsGradData, channels,
      imgSizeH, imgSizeW, sizeX, -pow, 2.0f * pow * scale, inputsGradData);
  CHECK_SYNC("KeCMRNormDiff");
}

}  // namespace paddle
