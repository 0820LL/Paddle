#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#define EIGEN_USE_GPU
#include "paddle/operators/clip_op.h"

#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename T>
__global__ void ClipGradientKernel(const int N, const T min, const T max,
                                   const T* Y, const T* dY, T* dX) {
  CUDA_1D_KERNEL_LOOP(i, N) { dX[i] = dY[i] * (Y[i] > min && Y[i] < max); }
}

template <typename T>
class ClipGradientOpCUDAKernel : public framework::OpKernel {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto max = context.op().GetAttr<float>("max");
    auto min = context.op().GetAttr<float>("min");
    auto* d_out = context.Input<Tensor>(framework::GradVarName("Out"));
    auto* d_x = context.Output<Tensor>(framework::GradVarName("X"));
    auto* x = context.Output<Tensor>("X");
    auto dims = d_x->dims();
    size_t count = 1;
    for (int i = 0; i < dims.size(); ++i) {
      count *= dims[i];
    }
    auto d_x_data = d_x->mutable_data<T>(context.GetPlace());
    auto d_out_data = d_out->data<T>();
    auto x_data = x->data<T>();

    int N = d_x->dims()[0];
    int D = d_x->dims()[1];
    int block = 512;
    int grid = (N * D + block - 1) / block;

    ClipGradientKernel<T><<<grid, block>>>(count, min, max, x_data, d_out_data,
                                           d_x_data);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_GPU_KERNEL(clip,
                       ops::ClipKernel<paddle::platform::GPUPlace, float>);
REGISTER_OP_GPU_KERNEL(clip_grad, ops::ClipGradientOpCUDAKernel<float>);
