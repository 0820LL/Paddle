#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#define EIGEN_USE_GPU
#include <stdio.h>
#include "paddle/operators/crop_op.h"

namespace paddle {
namespace operators {

using framework::LoDTensor;
using framework::Tensor;

template <typename T, int D>
__global__ void CropKernel(const int N, const int64_t* out_shape,
                           const int64_t* x_shape, const int* crop_rules,
                           const T* x_data, T* out_data) {
  int64_t pos[D];
  int tmp;
  int64_t x_index;
  for (int out_index = blockIdx.x * blockDim.x + threadIdx.x; out_index < N;
       out_index += blockDim.x * gridDim.x) {
    tmp = out_index;
    for (int64_t i = D - 1; i >= 0; --i) {
      pos[i] = (tmp % out_shape[i]) + crop_rules[i * 2];
      tmp = tmp / out_shape[i];
    }

    x_index = pos[0];
    for (size_t i = 1; i < D; ++i) {
      x_index = x_index * x_shape[i] + pos[i];
    }
    out_data[out_index] = x_data[x_index];
  }
}

template <typename T, int D>
void CropCUDAFunctoin(const framework::ExecutionContext& context) {
  PADDLE_ENFORCE(platform::is_gpu_place(context.GetPlace()),
                 "It must use GPUPlace.");
  auto* x = context.Input<LoDTensor>("X");
  auto* out = context.Output<LoDTensor>("Out");
  auto x_data = x->data<T>();
  T* out_data = out->mutable_data<T>(paddle::platform::GPUPlace());
  auto x_dims = x->dims();
  auto out_dims = out->dims();
  int64_t out_count = out->numel();
  Tensor x_shape;
  Tensor out_shape;
  int64_t* x_shape_data =
      x_shape.mutable_data<int64_t>({D}, paddle::platform::CPUPlace());
  int64_t* out_shape_data =
      out_shape.mutable_data<int64_t>({D}, paddle::platform::CPUPlace());
  for (int i = 0; i < D; ++i) {
    x_shape_data[i] = x_dims[i];
    out_shape_data[i] = out_dims[i];
  }
  Tensor x_shape_gpu;
  Tensor out_shape_gpu;
  x_shape_gpu.CopyFrom<int64_t>(x_shape, paddle::platform::GPUPlace());
  out_shape_gpu.CopyFrom<int64_t>(out_shape, paddle::platform::GPUPlace());
  auto offsets = context.op().Attr<std::vector<int>>("offsets");
  PADDLE_ENFORCE_EQ(
      D, offsets.size(),
      "Offsets size should be equal to dimension size of input tensor.");

  Tensor crop_rules;
  int* crop_rules_data =
      crop_rules.mutable_data<int>({D * 2}, paddle::platform::CPUPlace());
  for (size_t i = 0; i < D; ++i) {
    crop_rules_data[i * 2] = offsets[i];
    crop_rules_data[i * 2 + 1] = x_dims[i] - out_dims[i] - offsets[i];
  }

  Tensor crop_rules_gpu;
  crop_rules_gpu.CopyFrom<int>(crop_rules, paddle::platform::GPUPlace());

  int n = out_dims[0];
  int d = out_dims[1];
  int block = 512;
  int grid = (n * d + block - 1) / block;

  CropKernel<
      T,
      D><<<grid, block, 0, reinterpret_cast<const platform::CUDADeviceContext&>(
                               context.device_context())
                               .stream()>>>(
      out_count, out_shape_gpu.data<int64_t>(), x_shape_gpu.data<int64_t>(),
      crop_rules_gpu.data<int>(), x_data, out_data);
}

template <typename T>
class CropOpCUDAKernel : public framework::OpKernel {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    size_t rank = context.Input<LoDTensor>("X")->dims().size();
    switch (rank) {
      case 1:
        CropCUDAFunctoin<T, 1>(context);
        break;
      case 2:
        CropCUDAFunctoin<T, 2>(context);
        break;
      case 3:
        CropCUDAFunctoin<T, 3>(context);
        break;
      case 4:
        CropCUDAFunctoin<T, 4>(context);
        break;
      case 5:
        CropCUDAFunctoin<T, 5>(context);
        break;
      case 6:
        CropCUDAFunctoin<T, 6>(context);
        break;
      default:
        PADDLE_THROW(
            "CropOp only support tensors with no more than 6 dimensions.");
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_GPU_KERNEL(crop, ops::CropOpCUDAKernel<float>);
REGISTER_OP_GPU_KERNEL(crop_grad,
                       ops::CropGradKernel<paddle::platform::GPUPlace, float>);
