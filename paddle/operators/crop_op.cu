#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#define EIGEN_USE_GPU
#include "paddle/operators/crop_op.h"

#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename T, int D>
__global__ void CropKernel(const int N, const int64_t* out_shape,
                           const int64_t* x_shape, const int* crop_rules,
                           const T* x_data, T* out_data) {
  CUDA_1D_KERNEL_LOOP(index, N) {
    // int64_t dim_size = out_shape.size();
    int64_t pos[D];

    for (int64_t i = D - 1; i >= 0; --i) {
      pos[i] = (index % out_shape[i]) + crop_rules[i * 2];
      index = index / out_shape[i];
    }

    int64_t result = pos[0];
    for (size_t i = 1; i < D; ++i) {
      result = result * x_shape[i] + pos[i];
    }

    out_data[index] = x_data[result];
  }
}

template <typename T, int D>
void CropCUDAFunctoin(const framework::ExecutionContext& context) {
  auto* x = context.Input<Tensor>("X");
  auto* out = context.Output<Tensor>("Out");
  auto x_data = x->data<T>();
  T* out_data = out->mutable_data<T>(paddle::platform::CPUPlace());
  auto x_dims = x->dims();
  auto out_dims = out->dims();
  int64_t out_count = framework::product(out_dims);
  int64_t* x_shape = &(framework::vectorize(x_dims))[0];
  int64_t* out_shape = &(framework::vectorize(out_dims))[0];

  auto offsets = context.op().Attr<std::vector<int>>("offsets");
  PADDLE_ENFORCE_EQ(
      x_dims.size(), offsets.size(),
      "Offsets size should be equal to dimension size of input tensor.");

  int crop_rules[D * 2];
  for (size_t i = 0; i < x_dims.size(); ++i) {
    crop_rules[i * 2] = offsets[i];
    crop_rules[i * 2 + 1] = x_dims[i] - out_dims[i] - offsets[i];
  }

  int n = out_dims[0];
  int d = out_dims[1];
  int block = 512;
  int grid = (n * d + block - 1) / block;

  CropKernel<T, D><<<grid, block>>>(out_count, out_shape, x_shape, crop_rules,
                                    x_data, out_data);
}

template <typename T>
class CropOpCUDAKernel : public framework::OpKernel {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    size_t rank = context.Input<Tensor>("X")->dims().size();
    switch (rank) {
      case 1:
        CropCUDAFunctoin<T, 1>(context);
        break;
      case 2:
        CropCUDAFunctoin<T, 2>(context);
        break;
      case 3:
        CropCUDAFunctoin<T, 3>(context);
        break;
      case 4:
        CropCUDAFunctoin<T, 4>(context);
        break;
      case 5:
        CropCUDAFunctoin<T, 5>(context);
        break;
      case 6:
        CropCUDAFunctoin<T, 6>(context);
        break;
      default:
        PADDLE_THROW(
            "CropOp only support tensors with no more than 6 dimensions.");
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_GPU_KERNEL(crop, ops::CropOpCUDAKernel<float>);
REGISTER_OP_GPU_KERNEL(crop_grad,
                       ops::CropGradKernel<paddle::platform::GPUPlace, float>);
