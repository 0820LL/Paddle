#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#define EIGEN_USE_GPU
#include <stdio.h>
#include "paddle/operators/crop_op.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename T, int D>
__global__ void CropKernel(const int N, const int64_t* out_shape,
                           const int64_t* x_shape, const int* crop_rules,
                           const T* x_data, T* out_data) {
  int64_t pos[D];
  int tmp;
  int64_t x_index;
  for (int out_index = blockIdx.x * blockDim.x + threadIdx.x; out_index < N;
       out_index += blockDim.x * gridDim.x) {
    tmp = out_index;
    for (int64_t i = D - 1; i >= 0; --i) {
      pos[i] = (tmp % out_shape[i]) + crop_rules[i * 2];
      tmp = tmp / out_shape[i];
    }

    x_index = pos[0];
    for (size_t i = 1; i < D; ++i) {
      x_index = x_index * x_shape[i] + pos[i];
    }
    out_data[out_index] = x_data[x_index];
  }
}

template <typename T, int D>
void CropCUDAFunctoin(const framework::ExecutionContext& context) {
  PADDLE_ENFORCE(platform::is_gpu_place(context.GetPlace()),
                 "It must use GPUPlace.");
  LOG(INFO) << "CropCUDAFunctoin step1";
  auto* x = context.Input<Tensor>("X");
  auto* out = context.Output<Tensor>("Out");
  auto x_data = x->data<T>();
  T* out_data = out->mutable_data<T>(paddle::platform::GPUPlace());
  auto x_dims = x->dims();
  auto out_dims = out->dims();
  int64_t out_count = framework::product(out_dims);
  int64_t x_shape[D];
  int64_t out_shape[D];
  for (int i = 0; i < D; ++i) {
    x_shape[i] = x_dims[i];
    out_shape[i] = out_dims[i];
  }
  int64_t* x_shape_gpu;
  int64_t* out_shape_gpu;
  hipMalloc((void**)&x_shape_gpu, sizeof(int64_t) * D);
  hipMemcpy(x_shape_gpu, x_shape, sizeof(int64_t) * D, hipMemcpyHostToDevice);
  hipMalloc((void**)&out_shape_gpu, sizeof(int64_t) * D);
  hipMemcpy(out_shape_gpu, out_shape, sizeof(int64_t) * D,
             hipMemcpyHostToDevice);
  auto offsets = context.op().Attr<std::vector<int>>("offsets");
  PADDLE_ENFORCE_EQ(
      D, offsets.size(),
      "Offsets size should be equal to dimension size of input tensor.");

  int crop_rules[D * 2];
  for (size_t i = 0; i < x_dims.size(); ++i) {
    crop_rules[i * 2] = offsets[i];
    crop_rules[i * 2 + 1] = x_dims[i] - out_dims[i] - offsets[i];
  }

  int* crop_rules_gpu;
  hipMalloc((void**)&crop_rules_gpu, sizeof(int) * D * 2);
  hipMemcpy(crop_rules_gpu, crop_rules, sizeof(int) * D * 2,
             hipMemcpyHostToDevice);

  int n = out_dims[0];
  int d = out_dims[1];
  int block = 512;
  int grid = (n * d + block - 1) / block;
  CropKernel<T, D><<<grid, block>>>(out_count, out_shape_gpu, x_shape_gpu,
                                    crop_rules_gpu, x_data, out_data);
  hipFree(crop_rules_gpu);
  hipFree(x_shape_gpu);
  hipFree(out_shape_gpu);
}

template <typename T>
class CropOpCUDAKernel : public framework::OpKernel {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    size_t rank = context.Input<Tensor>("X")->dims().size();
    switch (rank) {
      case 1:
        CropCUDAFunctoin<T, 1>(context);
        break;
      case 2:
        CropCUDAFunctoin<T, 2>(context);
        break;
      case 3:
        CropCUDAFunctoin<T, 3>(context);
        break;
      case 4:
        CropCUDAFunctoin<T, 4>(context);
        break;
      case 5:
        CropCUDAFunctoin<T, 5>(context);
        break;
      case 6:
        CropCUDAFunctoin<T, 6>(context);
        break;
      default:
        PADDLE_THROW(
            "CropOp only support tensors with no more than 6 dimensions.");
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_GPU_KERNEL(crop, ops::CropOpCUDAKernel<float>);
REGISTER_OP_GPU_KERNEL(crop_grad,
                       ops::CropGradKernel<paddle::platform::GPUPlace, float>);
