/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
http://www.apache.org/licenseshashernless required by applicable law or agreed
to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#define EIGEN_USE_GPU
#include <functional>

#include "paddle/framework/lod_tensor.h"
#include "paddle/framework/op_registry.h"
#include "paddle/operators/nccl/nccl_gpu_common.h"

namespace paddle {
namespace operators {

using framework::Tensor;
using platform::Communicator;
using framework::LoDTensor;

template <typename Type>
class NCCLTypeWrapper;

template <>
class NCCLTypeWrapper<float> {
 public:
  static const ncclDataType_t type = ncclFloat;
};

template <>
class NCCLTypeWrapper<double> {
 public:
  static const ncclDataType_t type = ncclDouble;
};

template <typename T>
class NCCLAllReduceKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "This kernel only runs on GPU device.");

    auto ins = ctx.MultiInput<LoDTensor>("X");
    auto outs = ctx.MultiOutput<LoDTensor>("Out");

    auto* comm = ctx.Input<Communicator>("Communicator");

    auto stream = reinterpret_cast<const platform::CUDADeviceContext&>(
                      ctx.device_context())
                      .stream();
    // device id
    int device_id =
        boost::get<platform::GPUPlace>(ctx.GetPlace()).GetDeviceId();
    int idx = comm->GetCommId(device_id);

    for (size_t i = 0; i < ins.size(); ++i) {
      VLOG(1) << " invoke allreduce. send " << ins[i]->numel() << " recv "
              << outs[i]->numel();

      PADDLE_ENFORCE(platform::dynload::ncclAllReduce(
          ins[i]->data<T>(), outs[i]->mutable_data<T>(ctx.GetPlace()),
          outs[i]->numel(), NCCLTypeWrapper<T>::type, ncclSum,
          comm->comms_[idx], stream));
      PADDLE_ENFORCE(hipStreamSynchronize(stream));

      VLOG(1) << " finished allreduce. send " << ins[i]->numel() << " recv "
              << outs[i]->numel();
    }
  }
};

template <typename T>
class NCCLReduceKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "This kernel only runs on GPU device.");

    auto ins = ctx.MultiInput<LoDTensor>("X");  // x0, x1, x2
    auto outs = ctx.MultiOutput<LoDTensor>("Out");
    int root = ctx.Attr<int>("root");

    auto* comm = ctx.Input<Communicator>("Communicator");

    auto stream = reinterpret_cast<const platform::CUDADeviceContext&>(
                      ctx.device_context())
                      .stream();
    // device id
    int device_id =
        boost::get<platform::GPUPlace>(ctx.GetPlace()).GetDeviceId();
    int idx = comm->GetCommId(device_id);

    auto ins_names = ctx.Inputs("X");
    std::hash<std::string> hasher;
    for (size_t i = 0; i < ins.size(); ++i) {
      if (root == -1) {
        root = hasher(ins_names[i]) % comm->comms_.size();
      }
      T* recvbuffer = nullptr;
      if (root == device_id) {
        recvbuffer = outs[i]->mutable_data<T>(ctx.GetPlace());
      }

      VLOG(1) << " invoke reduce. send " << ins[i]->numel() << " recv "
              << outs[i]->numel();

      PADDLE_ENFORCE(platform::dynload::ncclReduce(
          ins[i]->data<T>(), recvbuffer, ins[i]->numel(),
          NCCLTypeWrapper<T>::type, ncclSum, root, comm->comms_[idx], stream));
      PADDLE_ENFORCE(hipStreamSynchronize(stream));

      VLOG(1) << " finished reduce. send " << ins[i]->numel() << " recv "
              << outs[i]->numel();
    }
  }
};

template <typename T>
class NCCLBcastKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "This kernel only runs on GPU device.");

    int root = ctx.Attr<int>("root");

    auto* comm = ctx.Input<Communicator>("Communicator");

    auto stream = reinterpret_cast<const platform::CUDADeviceContext&>(
                      ctx.device_context())
                      .stream();
    // device id
    int device_id =
        boost::get<platform::GPUPlace>(ctx.GetPlace()).GetDeviceId();
    int idx = comm->GetCommId(device_id);

    if (idx == root) {
      auto ins = ctx.MultiInput<LoDTensor>("X");
      for (size_t i = 0; i < ins.size(); ++i) {
        VLOG(1) << " invoke Bcast. send " << ins[i]->numel();

        PADDLE_ENFORCE(platform::dynload::ncclBcast(
            (void*)ins[i]->data<T>(), ins[i]->numel(), NCCLTypeWrapper<T>::type,
            root, comm->comms_[idx], stream));
        PADDLE_ENFORCE(hipStreamSynchronize(stream));

        VLOG(1) << " finished Bcast.";
      }
    } else {
      auto outs = ctx.MultiOutput<LoDTensor>("Out");
      for (size_t i = 0; i < outs.size(); ++i) {
        VLOG(1) << " invoke Bcast. recv. ";

        PADDLE_ENFORCE(platform::dynload::ncclBcast(
            outs[i]->mutable_data<T>(ctx.GetPlace()), outs[i]->numel(),
            NCCLTypeWrapper<T>::type, root, comm->comms_[idx], stream));
        PADDLE_ENFORCE(hipStreamSynchronize(stream));

        VLOG(1) << " finished Bcast. recv " << outs[i]->numel();
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_GPU_KERNEL(ncclAllReduce, ops::NCCLAllReduceKernel<float>);
REGISTER_OP_GPU_KERNEL(ncclBcast, ops::NCCLBcastKernel<float>);
REGISTER_OP_GPU_KERNEL(ncclReduce, ops::NCCLReduceKernel<float>);
