/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
http://www.apache.org/licenseshashernless required by applicable law or agreed
to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#define EIGEN_USE_GPU
#include <functional>

#include "paddle/operators/nccl_op.h"

namespace paddle {
namespace operators {

template <typename T>
class NCCLAllReduceKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "This kernel only runs on GPU device.");

    auto ins = ctx.MultiInput<Tensor>("X");
    auto outs = ctx.MultiOutput<Tensor>("Out");

    auto* comm = ctx.Input<Communicator>("Communicator");

    auto stream = reinterpret_cast<const platform::CUDADeviceContext&>(
                      ctx.device_context())
                      .stream();
    // device id
    int device_id =
        boost::get<platform::GPUPlace>(ctx.GetPlace()).GetDeviceId();
    int idx = comm->GetCommId(device_id);

    for (size_t i = 0; i < ins.size(); ++i) {
      PADDLE_ENFORCE(platform::dynload::ncclAllReduce(
          ins[i]->data<T>(), outs[i]->mutable_data<T>(ctx.GetPlace()),
          outs[i]->numel() * sizeof(T), NCCLTypeWrapper<T>::type, ncclSum,
          comm->comms_[idx], stream));
      PADDLE_ENFORCE(hipStreamSynchronize(stream));
    }
  }
};

template <typename T>
class NCCLReduceKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "This kernel only runs on GPU device.");

    auto ins = ctx.MultiInput<Tensor>("X");  // x0, x1, x2
    auto outs = ctx.MultiOutput<Tensor>("Out");

    auto* comm = ctx.Input<Communicator>("Communicator");

    auto stream = reinterpret_cast<const platform::CUDADeviceContext&>(
                      ctx.device_context())
                      .stream();
    // device id
    int device_id =
        boost::get<platform::GPUPlace>(ctx.GetPlace()).GetDeviceId();
    int idx = comm->GetCommId(device_id);

    auto ins_names = ctx.Inputs("X");
    std::hash<std::string> hasher;
    for (size_t i = 0; i < ins.size(); ++i) {
      int root = hasher(ins_names[i]) % comm->comms_.size();
      T* recvbuffer = nullptr;
      if (root == device_id) {
        recvbuffer = outs[i]->mutable_data<T>(ctx.GetPlace());
      }
      PADDLE_ENFORCE(platform::dynload::ncclReduce(
          ins[i]->data<T>(), recvbuffer, ins[i]->numel(),
          NCCLTypeWrapper<T>::type, ncclSum, root, comm->comms_[idx], stream));
      PADDLE_ENFORCE(hipStreamSynchronize(stream));
    }
  }
};

template <typename T>
class NCCLBcastKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "This kernel only runs on GPU device.");

    int root = ctx.Attr<int>("root");

    auto* comm = ctx.Input<Communicator>("Communicator");

    auto stream = reinterpret_cast<const platform::CUDADeviceContext&>(
                      ctx.device_context())
                      .stream();
    // device id
    int device_id =
        boost::get<platform::GPUPlace>(ctx.GetPlace()).GetDeviceId();
    int idx = comm->GetCommId(device_id);
    if (idx == root) {
      auto ins = ctx.MultiInput<Tensor>("X");
      for (size_t i = 0; i < ins.size(); ++i) {
        PADDLE_ENFORCE(platform::dynload::ncclBcast(
            (void*)ins[i]->data<T>(), ins[i]->numel(), NCCLTypeWrapper<T>::type,
            root, comm->comms_[idx], stream));
        PADDLE_ENFORCE(hipStreamSynchronize(stream));
      }
    } else {
      auto outs = ctx.MultiOutput<Tensor>("Out");
      for (size_t i = 0; i < outs.size(); ++i) {
        PADDLE_ENFORCE(platform::dynload::ncclBcast(
            outs[i]->mutable_data<T>(ctx.GetPlace()), outs[i]->numel(),
            NCCLTypeWrapper<T>::type, root, comm->comms_[idx], stream));
        PADDLE_ENFORCE(hipStreamSynchronize(stream));
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_GPU_KERNEL(ncclAllReduce, ops::NCCLAllReduceKernel<float>);
REGISTER_OP_GPU_KERNEL(ncclBcastSend, ops::NCCLBcastKernel<float>);
REGISTER_OP_GPU_KERNEL(ncclReduce, ops::NCCLReduceKernel<float>);
REGISTER_OP_GPU_KERNEL(ncclBcastRecv, ops::NCCLBcastKernel<float>);
