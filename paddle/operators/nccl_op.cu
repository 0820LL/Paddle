/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#define EIGEN_USE_GPU
#include "paddle/operators/nccl_op.h"

namespace paddle {
namespace operators {

template <typename T>
class NCCLAllReduceKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "This kernel only runs on GPU device.");

    auto ins = ctx.MultiInput<Tensor>("X");
    auto outs = ctx.MultiOutput<Tensor>("Out");
    std::string reduction = ctx.Attr<std::string>("reduction");
    ncclRedOp_t op_type;
    if (reduction == "ncclSum") {
      op_type = ncclSum;
    } else if (reduction == "ncclProd") {
      op_type = ncclProd;
    } else if (reduction == "ncclMin") {
      op_type = ncclMin;
    } else if (reduction == "ncclMax") {
      op_type = ncclMax;
    } else {
      PADDLE_ENFORCE(false, "reduction error.");
    }

    auto* comm = ctx.Input<Communicator>("Communicator");

    auto stream = reinterpret_cast<const platform::CUDADeviceContext&>(
                      ctx.device_context())
                      .stream();

    // device id
    int device_id =
        boost::get<platform::GPUPlace>(ctx.GetPlace()).GetDeviceId();
    int idx = comm->GetCommId(device_id);

    for (size_t i = 0; i < ins.size(); ++i) {
      PADDLE_ENFORCE(ncclAllReduce(
          ins[i]->data<T>(), outs[i]->mutable_data<T>(ctx.GetPlace()),
          outs[i]->numel() * sizeof(T), NCCLTypeWrapper<T>::type, op_type,
          comm->comms_[idx], stream));
      PADDLE_ENFORCE(hipStreamSynchronize(stream));
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_GPU_KERNEL(ncclAllReduce, ops::NCCLAllReduceKernel<float>);
