#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include "paddle/framework/op_registry.h"
#include "paddle/operators/cross_entropy_op.h"
#include "paddle/platform/assert.h"
#include "paddle/platform/hostdevice.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void CrossEntropyKernel(T* Y, const T* X, const int* label,
                                   const int N, const int D) {
  // TOOD(qingqing) define CUDA_1D_KERNEL_LOOP macro in a common file.
  // CUDA_1D_KERNEL_LOOP(i, N) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    PADDLE_ASSERT(label[i] >= 0 && label[i] < D);
    Y[i] = -TolerableValue<T>()(log(X[i * D + label[i]]));
  }
}

template <typename T, int BlockSize>
__global__ void SoftCrossEntropyKernel(T* Y, const T* X, const T* label,
                                       const int N, const int D) {
  int tid = threadIdx.x;
  __shared__ T d_sum[BlockSize];
  int next_idx = blockIdx.x * D + tid;

  d_sum[tid] = 0;
  int cur_idx = tid;
  while (cur_idx < D) {
    d_sum[tid] += TolerableValue<T>()(std::log(X[next_idx])) * label[next_idx];
    next_idx += BlockSize;
    cur_idx += BlockSize;
  }
  __syncthreads();

  for (int stride = BlockSize >> 1; stride > 0; stride >>= 1) {
    __syncthreads();
    if (tid < stride) {
      next_idx = tid + stride;
      d_sum[tid] += d_sum[next_idx];
    }
  }
  __syncthreads();

  if (tid == 0) {
    Y[blockIdx.x] = -d_sum[0];
  }
}

// TODO(qingqing): make zero setting an common function.
template <typename T>
__global__ void zero(T* X, const int N) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    X[i] = 0.0;
  }
}

template <typename T>
__global__ void CrossEntropyGradientKernel(T* dX, const T* dY, const T* X,
                                           const int* label, const int N,
                                           const int D) {
  // TOOD(qingqing) define CUDA_1D_KERNEL_LOOP macro in a common file.
  // CUDA_1D_KERNEL_LOOP(i, N) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    int idx = i * D + label[i];
    dX[idx] = -dY[i] / X[idx];
  }
}

template <typename T>
__global__ void SoftCrossEntropyGradientKernel(T* dX, const T* dY, const T* X,
                                               const T* label, const int N,
                                               const int D) {
  int row_ids = blockIdx.x * blockDim.x + threadIdx.x;
  int col_ids = blockIdx.y * blockDim.y + threadIdx.y;
  int ids = row_ids * D + col_ids;

  if (ids < N * D) {
    dX[ids] = -label[ids] * dY[row_ids] / X[ids];
  }
}

template <typename T>
class CrossEntropyOpCUDAKernel : public framework::OpKernel {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "This kernel only runs on GPU device.");

    auto x = ctx.Input<Tensor>("X");
    auto y = ctx.Output<Tensor>("Y");
    auto label = ctx.Input<Tensor>("Label");

    auto* x_data = x->data<T>();
    y->mutable_data<T>(ctx.GetPlace());
    auto* y_data = y->data<T>();

    int n = x->dims()[0];
    int d = x->dims()[1];
    int block = 512;
    int grid = (n + block - 1) / block;
    // TODO(qingqing) launch kernel on specified stream
    // base on ExecutionContext.
    if (ctx.Attr<int>("soft_label") == 1) {
      auto* label_data = ctx.Input<Tensor>("Label")->data<T>();
      grid = d;
      SoftCrossEntropyKernel<T, 512><<<grid, block>>>(y_data, x_data,
                                                      label_data, n, d);
    } else {
      auto* label_data = ctx.Input<Tensor>("Label")->data<int>();
      CrossEntropyKernel<T><<<grid, block>>>(y_data, x_data, label_data, n, d);
    }
  }
};

template <typename T>
class CrossEntropyGradientOpCUDAKernel : public framework::OpKernel {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "This kernel only runs on GPU device.");

    auto x = ctx.Input<Tensor>("X");
    auto dx = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto dy = ctx.Input<Tensor>(framework::GradVarName("Y"));
    auto label = ctx.Input<Tensor>("Label");

    auto* dx_data = dx->mutable_data<T>(ctx.GetPlace());
    auto* dy_data = dy->data<T>();
    auto* x_data = x->data<T>();

    int n = x->dims()[0];
    int d = x->dims()[1];
    int block = 512;
    int grid = (n * d + block - 1) / block;
    zero<T><<<grid, block>>>(dx_data, n * d);
    grid = (n + block - 1) / block;
    // TODO(qingqing): launch kernel on specified stream
    // base on ExecutionContext.
    if (ctx.Attr<int>("soft_label") == 1) {
      int block_x = 32;
      int block_y = 32;
      dim3 block(block_x, block_y);
      dim3 grid((n + block_x - 1) / block_x, (d + block_y - 1) / block_y);

      auto* label_data = label->data<T>();
      SoftCrossEntropyGradientKernel<T><<<grid, block>>>(
          dx_data, dy_data, x_data, label_data, n, d);
    } else {
      auto* label_data = label->data<int>();
      CrossEntropyGradientKernel<T><<<grid, block>>>(dx_data, dy_data, x_data,
                                                     label_data, n, d);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_GPU_KERNEL(cross_entropy, ops::CrossEntropyOpCUDAKernel<float>);
REGISTER_OP_GPU_KERNEL(cross_entropy_grad,
                       ops::CrossEntropyGradientOpCUDAKernel<float>);
