#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/operators/math/math_function.h"

namespace paddle {
namespace operators {
namespace math {

template <>
void gemm<platform::GPUPlace, float>(const platform::DeviceContext& context,
                                     const CBLAS_TRANSPOSE transA,
                                     const CBLAS_TRANSPOSE transB, const int M,
                                     const int N, const int K,
                                     const float alpha, const float* A,
                                     const float* B, const float beta,
                                     float* C) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  int lda = (transA == CblasNoTrans) ? K : M;
  int ldb = (transB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (transA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (transB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

  PADDLE_ENFORCE(platform::dynload::hipblasSgemm(
      reinterpret_cast<const platform::CUDADeviceContext&>(context)
          .cublas_handle(),
      cuTransB, cuTransA, N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void gemm<platform::GPUPlace, double>(const platform::DeviceContext& context,
                                      const CBLAS_TRANSPOSE transA,
                                      const CBLAS_TRANSPOSE transB, const int M,
                                      const int N, const int K,
                                      const double alpha, const double* A,
                                      const double* B, const double beta,
                                      double* C) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  int lda = (transA == CblasNoTrans) ? K : M;
  int ldb = (transB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (transA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (transB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  PADDLE_ENFORCE(platform::dynload::hipblasDgemm(
      reinterpret_cast<const platform::CUDADeviceContext&>(context)
          .cublas_handle(),
      cuTransB, cuTransA, N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void gemm<platform::GPUPlace, float>(const platform::DeviceContext& context,
                                     const bool transA, const bool transB,
                                     const int M, const int N, const int K,
                                     const float alpha, const float* A,
                                     const int lda, const float* B,
                                     const int ldb, const float beta, float* C,
                                     const int ldc) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  hipblasOperation_t cuTransA = transA == false ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB = transB == false ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  PADDLE_ENFORCE(platform::dynload::hipblasSgemm(
      reinterpret_cast<const platform::CUDADeviceContext&>(context)
          .cublas_handle(),
      cuTransB, cuTransA, N, M, K, &alpha, B, ldb, A, lda, &beta, C, ldc));
}

template <>
void gemm<platform::GPUPlace, double>(const platform::DeviceContext& context,
                                      const bool transA, const bool transB,
                                      const int M, const int N, const int K,
                                      const double alpha, const double* A,
                                      const int lda, const double* B,
                                      const int ldb, const double beta,
                                      double* C, const int ldc) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  hipblasOperation_t cuTransA = transA == false ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB = transB == false ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  PADDLE_ENFORCE(platform::dynload::hipblasDgemm(
      reinterpret_cast<const platform::CUDADeviceContext&>(context)
          .cublas_handle(),
      cuTransB, cuTransA, N, M, K, &alpha, B, ldb, A, lda, &beta, C, ldc));
}

template <>
void matmul<platform::GPUPlace, float>(
    const platform::DeviceContext& context, const framework::Tensor& matrix_a,
    bool trans_a, const framework::Tensor& matrix_b, bool trans_b, float alpha,
    framework::Tensor* matrix_out, float beta) {
  auto dim_a = matrix_a.dims();
  auto dim_b = matrix_b.dims();
  auto dim_out = matrix_out->dims();
  PADDLE_ENFORCE(dim_a.size() == 2 && dim_b.size() == 2 && dim_out.size() == 2,
                 "The input and output of matmul be matrix");

  PADDLE_ENFORCE(platform::is_gpu_place(matrix_a.place()) &&
                     platform::is_gpu_place(matrix_b.place()) &&
                     platform::is_gpu_place(matrix_out->place()),
                 "Matrix must all be in GPUPlace");

  int M = dim_out[0];
  int N = dim_out[1];
  int K = (trans_a == false) ? dim_a[1] : dim_a[0];

  CBLAS_TRANSPOSE transA = (trans_a == false) ? CblasNoTrans : CblasTrans;
  CBLAS_TRANSPOSE transB = (trans_b == false) ? CblasNoTrans : CblasTrans;

  gemm<platform::GPUPlace, float>(
      context, transA, transB, M, N, K, alpha, matrix_a.data<float>(),
      matrix_b.data<float>(), beta, matrix_out->data<float>());
}

template <>
void matmul<platform::GPUPlace, double>(
    const platform::DeviceContext& context, const framework::Tensor& matrix_a,
    bool trans_a, const framework::Tensor& matrix_b, bool trans_b, double alpha,
    framework::Tensor* matrix_out, double beta) {
  auto dim_a = matrix_a.dims();
  auto dim_b = matrix_b.dims();
  auto dim_out = matrix_out->dims();
  PADDLE_ENFORCE(dim_a.size() == 2 && dim_b.size() == 2 && dim_out.size() == 2,
                 "The input and output of matmul be matrix");

  PADDLE_ENFORCE(platform::is_gpu_place(matrix_a.place()) &&
                     platform::is_gpu_place(matrix_b.place()) &&
                     platform::is_gpu_place(matrix_out->place()),
                 "Matrix must all be in GPUPlace");

  int M = dim_out[0];
  int N = dim_out[1];
  int K = (trans_a == false) ? dim_a[1] : dim_a[0];

  CBLAS_TRANSPOSE transA = (trans_a == false) ? CblasNoTrans : CblasTrans;
  CBLAS_TRANSPOSE transB = (trans_b == false) ? CblasNoTrans : CblasTrans;

  gemm<platform::GPUPlace, double>(
      context, transA, transB, M, N, K, alpha, matrix_a.data<double>(),
      matrix_b.data<double>(), beta, matrix_out->data<double>());
}

template <typename T>
struct SelectedRowsAdd<platform::GPUPlace, T> {
  void operator()(const platform::DeviceContext& context,
                  const framework::SelectedRows& input1,
                  const framework::SelectedRows& input2,
                  framework::SelectedRows* output) {
    auto in1_height = input1.height();
    PADDLE_ENFORCE_EQ(in1_height, input2.height());
    output->set_height(in1_height);

    auto& in1_rows = input1.rows();
    auto& in2_rows = input2.rows();
    std::vector<int64_t> out_rows;
    out_rows.reserve(in1_rows.size() + in2_rows.size());

    // concat rows
    out_rows.insert(out_rows.end(), in1_rows.begin(), in1_rows.end());
    out_rows.insert(out_rows.end(), in2_rows.begin(), in2_rows.end());
    output->set_rows(out_rows);

    auto* out_value = output->mutable_value();
    auto& in1_value = input1.value();
    auto& in2_value = input2.value();

    auto in1_row_numel = in1_value.numel() / in1_rows.size();
    PADDLE_ENFORCE_EQ(in1_row_numel, in2_value.numel() / in2_rows.size());
    PADDLE_ENFORCE_EQ(in1_row_numel, out_value->numel() / out_rows.size());

    auto* out_data = out_value->data<T>();
    auto* in1_data = in1_value.data<T>();

    auto in1_place = input1.place();
    PADDLE_ENFORCE(platform::is_gpu_place(in1_place));
    auto in2_place = input2.place();
    PADDLE_ENFORCE(platform::is_gpu_place(in2_place));
    auto out_place = context.GetPlace();
    PADDLE_ENFORCE(platform::is_gpu_place(out_place))

    memory::Copy(
        boost::get<platform::GPUPlace>(out_place), out_data,
        boost::get<platform::GPUPlace>(in1_place), in1_data,
        in1_value.numel() * sizeof(T),
        reinterpret_cast<const platform::CUDADeviceContext&>(context).stream());

    auto* in2_data = in2_value.data<T>();
    memory::Copy(
        boost::get<platform::GPUPlace>(out_place), out_data + in1_value.numel(),
        boost::get<platform::GPUPlace>(in2_place), in2_data,
        in2_value.numel() * sizeof(T),
        reinterpret_cast<const platform::CUDADeviceContext&>(context).stream());
  }
};

template struct SelectedRowsAdd<platform::GPUPlace, float>;

namespace {
template <int block_size, typename T>
__global__ void SelectedRowsAddTensorKernel(T* selected_rows, int64_t* rows,
                                            T* tensor_in, T* tensor_out,
                                            const int64_t row_numel) {
  const ty = blockIdx.y;
  int tid = threadIdx.x;

  selected_rows += ty * row_numel;
  tensor_in += rows[ty] * row_numel;
  tensor_out += rows[ty] * row_numel;

  for (int index = tid; index < row_numel; index += block_size) {
    tensor_out[index] = tensor_in[index] + selected_rows[index];
  }
}
}

template <typename T>
struct SelectedRowsAddTensor<platform::GPUPlace, T> {
  void operator()(const platform::DeviceContext& context,
                  const framework::SelectedRows& input1,
                  const framework::Tensor& input2, framework::Tensor* output) {
    auto in1_height = input1.height();
    auto in2_dims = input2.dims();
    auto out_dims = output->dims();
    PADDLE_ENFORCE_EQ(in1_height, in2_dims[0]);
    PADDLE_ENFORCE_EQ(in1_height, out_dims[0]);

    auto& in1_value = input1.value();
    auto& in1_rows = input1.rows();

    int64_t in1_row_numel = in1_value.numel() / in1_rows.size();
    PADDLE_ENFORCE_EQ(in1_row_numel, input2.numel() / in1_height);
    PADDLE_ENFORCE_EQ(in1_row_numel, output->numel() / in1_height);

    auto* in1_data = in1_value.data<T>();
    auto* in2_data = input2.data<T>();
    auto* out_data = output->data<T>();

    const int block_size = 256;
    dim3 threads(block_size, 1);
    dim3 grid(1, in1_height);
    SelectedRowsAddTensorKernel<block_size, T><<<
        grid, threads, 0,
        reinterpret_cast<const platform::CUDADeviceContext&>(ctx).stream()>>>(
        in1_data, in1_rows.data(), in2_data, out_data, in1_row_numel);
  }
};

template struct SelectedRowsAddTensor<platform::GPUPlace, float>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
