#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include "paddle/memory/memcpy.h"
#include "paddle/memory/memory.h"
#include "paddle/operators/transpose_op.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void transpose_kernel(int nthreads, const T* in_data, T* out_data,
                                 int* offset_buffer, int ndims) {
  int* in_offset = offset_buffer;
  int* out_offset = offset_buffer + ndims;
  int* axis = offset_buffer + ndims;

  int to_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (to_index < nthreads) {
    int from_index = 0;
    int temp = to_index;
    for (size_t i = 0; i < ndims; i++) {
      from_index += (temp / out_offset[i]) * in_offset[axis[i]];
      temp = temp % out_offset[i];
    }
    out_data[to_index] = in_data[from_index];
  }
}

template <typename T>
void TransposeCUDA(const framework::ExecutionContext& context,
                   const framework::Tensor& in, framework::Tensor& out,
                   std::vector<int> axis) {
  auto* in_data = in.template data<T>();
  auto* out_data = out.template mutable_data<T>(context.GetPlace());
  auto in_dim = in.dims();
  auto out_dim = out.dims();
  auto data_size = product(in_dim);
  size_t ndims = in_dim.size();
  std::vector<int> in_offset(ndims, 1);
  std::vector<int> out_offset(ndims, 1);
  std::vector<int64_t> buffer_dim_shape(1, ndims * 3);

  auto buffer_dims = framework::make_ddim(buffer_dim_shape);
  framework::Tensor host_buffer;
  platform::CPUPlace cpu_place;
  platform::GPUPlace gpu_place;

  int* host_buffer_data = host_buffer.mutable_data<int>(buffer_dims, cpu_place);

  auto offset_buffer =
      memory::Alloc(context.GetPlace(), ndims * 3 * sizeof(int));

  for (int i = ndims - 2; i >= 0; i--) {
    in_offset[i] = in_offset[i + 1] * in_dim[i + 1];
    out_offset[i] = out_offset[i + 1] * out_dim[i + 1];
  }

  for (int i = 0; i < ndims; i++) {
    host_buffer_data[i] = in_offset[i];
    host_buffer_data[i + ndims] = out_offset[i];
    host_buffer_data[i + ndims * 2] = axis[i];
  }

  memory::Copy(gpu_place, offset_buffer, cpu_place, host_buffer_data,
               ndims * 3 * sizeof(int));
  int block = 512;
  int grid = (data_size + block - 1) / block;
  transpose_kernel<T><<<grid, block>>>(data_size, in_data, out_data,
                                       static_cast<int*>(offset_buffer), ndims);
  memory::Free(gpu_place, offset_buffer);
}

template <typename T>
class TransposeCUDAKernel : public framework::OpKernel {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(context.GetPlace()),
                   "It must use GPUPlace.");
    auto* in = context.Input<framework::Tensor>("X");
    auto* out = context.Output<framework::Tensor>("Out");
    auto axis = context.GetAttr<std::vector<int>>("axis");
    TransposeCUDA<T>(context, *in, *out, axis);
  }
};

template <typename T>
class TransposeGradCUDAKernel : public framework::OpKernel {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(context.GetPlace()),
                   "It must use GPUPlace.");
    auto* in = context.Input<framework::Tensor>(framework::GradVarName("Out"));
    auto* out = context.Output<framework::Tensor>(framework::GradVarName("X"));
    auto axis_temp = context.GetAttr<std::vector<int>>("axis");

    std::vector<int> axis(axis_temp);

    for (size_t i = 0; i < axis.size(); i++) {
      axis[axis_temp[i]] = i;
    }
    TransposeCUDA<T>(context, *in, *out, axis);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_GPU_KERNEL(transpose, ops::TransposeCUDAKernel<float>);
REGISTER_OP_GPU_KERNEL(transpose_grad, ops::TransposeGradCUDAKernel<float>);
