#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#define EIGEN_USE_GPU

#include "paddle/framework/op_registry.h"
#include "paddle/operators/cross_entropy_op.h"
#include "paddle/operators/math/softmax.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename T>
__global__ void CrossEntropy(T* out, const T* softmax_out, const int* labels,
                             const int batch_size, const int class_num) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < batch_size) {
    PADDLE_ASSERT(labels[i] >= 0 && labels[i] < class_num);
    out[i] =
        -TolerableValue<T>()(std::log(softmax_out[i * class_num + labels[i]]));
  }
}

template <typename T>
__global__ void CrossEntropyGrad(T* out_grad, const T* in_grad,
                                 const int* labels, const int batch_size,
                                 const int class_num) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int sample_idx = tid / class_num;

  if (tid < batch_size * class_num) out_grad[tid] *= in_grad[sample_idx];
  __syncthreads();

  if (tid < batch_size) {
    PADDLE_ASSERT(labels[sample_idx] >= 0 && labels[sample_idx] < class_num);
    out_grad[tid * class_num + labels[tid]] -= 1.;
  }
}

template <typename T>
__device__ __forceinline__ T sum_single_warp(T val) {
  val += __shfl_down(val, 16);
  val += __shfl_down(val, 8);
  val += __shfl_down(val, 4);
  val += __shfl_down(val, 2);
  val += __shfl_down(val, 1);
  return val;
}

template <typename T>
__global__ void SoftCrossEntropyKernel(T* Y, const T* X, const T* label,
                                       const int class_num) {
  int tid = threadIdx.x;
  extern __shared__ T d_sum[];
  d_sum[tid] = 0;

  int cur_idx = tid;
  int next_idx = blockIdx.x * class_num + tid;
  while (cur_idx < class_num) {
    d_sum[tid] += TolerableValue<T>()(std::log(X[next_idx])) * label[next_idx];
    next_idx += blockDim.x;
    cur_idx += blockDim.x;
  }
  __syncthreads();

  for (unsigned int stride = blockDim.x >> 1; stride >= 32; stride >>= 1) {
    if (tid < stride) d_sum[tid] += d_sum[tid + stride];
    __syncthreads();
  }

  T val = d_sum[tid];
  val = sum_single_warp<T>(val);
  if (tid == 0) Y[blockIdx.x] = -val;
}

template <typename T>
__global__ void SoftCrossEntropyGradientKernel(T* logit_grad,
                                               const T* loss_grad,
                                               const T* labels,
                                               const int batch_size,
                                               const int class_num) {
  int ids = blockIdx.x * blockDim.x + threadIdx.x;
  if (ids < batch_size * class_num) {
    int row_ids = ids / class_num;
    logit_grad[ids] = logit_grad[ids] * loss_grad[row_ids] - labels[ids];
  }
}

template <typename T>
class SoftmaxWithCrossEntropyCUDAKernel : public framework::OpKernel {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(context.GetPlace()),
                   "This kernel only runs on GPU device.");
    T* loss_data =
        context.Output<Tensor>("Loss")->mutable_data<T>(context.GetPlace());

    const Tensor* logits = context.Input<Tensor>("Logits");
    Tensor* softmax = context.Output<Tensor>("Softmax");
    T* softmax_out = softmax->mutable_data<T>(context.GetPlace());
    math::SoftmaxFunctor<platform::GPUPlace, T>()(context, logits, softmax);

    const int batch_size = logits->dims()[0];
    const int class_num = logits->dims()[1];
    int block = 512;
    int grid = (batch_size + block - 1) / block;

    if (context.Attr<bool>("softLabel")) {
      const T* label_data = context.Input<Tensor>("Label")->data<T>();
      block = class_num > 512 ? 512 : pow(2, int(std::log2(class_num)));

      SoftCrossEntropyKernel<
          T><<<batch_size, block, block * sizeof(T),
               reinterpret_cast<const platform::CUDADeviceContext&>(
                   context.device_context())
                   .stream()>>>(loss_data, softmax_out, label_data, class_num);
    } else {
      const int* label_data = context.Input<Tensor>("Label")->data<int>();
      CrossEntropy<T><<<grid, block, 0,
                        reinterpret_cast<const platform::CUDADeviceContext&>(
                            context.device_context())
                            .stream()>>>(loss_data, softmax_out, label_data,
                                         batch_size, class_num);
    }
  }
};

template <typename T>
class SoftmaxWithCrossEntropyGradCUDAKernel : public framework::OpKernel {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(context.GetPlace()),
                   "This kernel only runs on GPU device.");
    const Tensor* labels = context.Input<Tensor>("Label");
    const T* loss_grad_data =
        context.Input<Tensor>(framework::GradVarName("Loss"))->data<T>();
    Tensor* logit_grad =
        context.Output<Tensor>(framework::GradVarName("Logits"));
    logit_grad->ShareDataWith<T>(*context.Input<Tensor>("Softmax"));
    T* logit_grad_data = logit_grad->data<T>();

    const int batch_size = logit_grad->dims()[0];
    const int class_num = logit_grad->dims()[1];
    int block = 512;
    int grid = (batch_size * class_num + block - 1) / block;

    if (context.Attr<bool>("softLabel")) {
      const T* label_data = labels->data<T>();
      SoftCrossEntropyGradientKernel<T><<<
          grid, block, 0, reinterpret_cast<const platform::CUDADeviceContext&>(
                              context.device_context())
                              .stream()>>>(logit_grad_data, loss_grad_data,
                                           label_data, batch_size, class_num);
    } else {
      const int* label_data = labels->data<int>();
      CrossEntropyGrad<T><<<
          grid, block, 0, reinterpret_cast<const platform::CUDADeviceContext&>(
                              context.device_context())
                              .stream()>>>(logit_grad_data, loss_grad_data,
                                           label_data, batch_size, class_num);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_GPU_KERNEL(softmax_with_cross_entropy,
                       ops::SoftmaxWithCrossEntropyCUDAKernel<float>);
REGISTER_OP_GPU_KERNEL(softmax_with_cross_entropy_grad,
                       ops::SoftmaxWithCrossEntropyGradCUDAKernel<float>);
