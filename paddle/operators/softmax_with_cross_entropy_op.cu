#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#define EIGEN_USE_GPU

#include "paddle/framework/op_registry.h"
#include "paddle/operators/cross_entropy_op.h"
#include "paddle/operators/math/softmax.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename T>
__global__ void CrossEntropyKernel(T* out, const T* softmax_out,
                                   const int* label, const int batch_size,
                                   const int class_num) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < batch_size) {
    PADDLE_ASSERT(label[i] >= 0 && label[i] < class_num);
    out[i] = -tolerable_value(std::log(softmax_out[i * class_num + label[i]]));
  }
}

template <typename T>
__global__ void CrossEntropyWithSoftmaxGradKernel(T* softmax_out,
                                                  const int* label,
                                                  const int batch_size,
                                                  const int class_num) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < batch_size) {
    PADDLE_ASSERT(label[i] >= 0 && label[i] < class_num);
    softmax_out[i * class_num + label[i]] -= 1.;
  }
}

template <typename T>
class SoftmaxWithCrossEntropyCUDAKernel : public framework::OpKernel {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(context.GetPlace()),
                   "This kernel only runs on GPU device.");

    // Calculate ths softmax outputs.
    const Tensor* logits = context.Input<Tensor>("Logits");
    Tensor* softmax = context.Output<Tensor>("Softmax");
    softmax->mutable_data<T>(context.GetPlace());
    math::SoftmaxFunctor<platform::GPUPlace, T>()(logits, softmax, context);
    T* softmax_out = softmax->data<T>();

    // Calculate the cross entropy loss based on hard labels.
    const int* label_data = context.Input<Tensor>("Label")->data<int>();
    Tensor* loss = context.Output<Tensor>("Loss");
    loss->mutable_data<T>(context.GetPlace());
    T* loss_data = loss->data<T>();

    const int batch_size = logits->dims()[0];
    const int class_num = logits->dims()[1];
    int block = 512;
    int grid = (batch_size + block - 1) / block;

    CrossEntropyKernel<T><<<grid, block>>>(loss_data, softmax_out, label_data,
                                           batch_size, class_num);
  }
};

template <typename T>
class SoftmaxWithCrossEntropyGradCUDAKernel : public framework::OpKernel {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(context.GetPlace()),
                   "This kernel only runs on GPU device.");

    Tensor* logit_grad =
        context.Output<Tensor>(framework::GradVarName("Logits"));
    logit_grad->ShareDataWith<T>(*context.Input<Tensor>("Softmax"));
    T* logit_grad_data = logit_grad->data<T>();

    const int batch_size = logit_grad->dims()[0];
    const int class_num = logit_grad->dims()[1];

    const int* label_data = context.Input<Tensor>("Label")->data<int>();

    const int block = 512;
    const int grid = (batch_size + block - 1) / block;

    CrossEntropyWithSoftmaxGradKernel<T><<<grid, block>>>(
        logit_grad_data, label_data, batch_size, class_num);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_GPU_KERNEL(softmax_with_cross_entropy,
                       ops::SoftmaxWithCrossEntropyCUDAKernel<float>);
REGISTER_OP_GPU_KERNEL(softmax_with_cross_entropy_grad,
                       ops::SoftmaxWithCrossEntropyGradCUDAKernel<float>);
